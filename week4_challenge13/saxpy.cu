#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, const float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(int argc, char **argv)
{
    int pow = (argc > 1) ? atoi(argv[1]) : 20;   // default N = 2^20
    int N   = 1 << pow;
    size_t bytes = N * sizeof(float);

    // Host buffers
    float *h_x = (float*)malloc(bytes);
    float *h_y = (float*)malloc(bytes);
    for (int i = 0; i < N; ++i) { h_x[i] = 1.0f; h_y[i] = 2.0f; }

    // Device buffers
    float *d_x, *d_y;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);

    hipEvent_t start, stop;
    float h2d_ms, kernel_ms, d2h_ms;
    hipEventCreate(&start); hipEventCreate(&stop);

    // ---------------- Host ➜ Device copy ----------------
    hipEventRecord(start);
    hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, bytes, hipMemcpyHostToDevice);
    hipEventRecord(stop);  hipEventSynchronize(stop);
    hipEventElapsedTime(&h2d_ms, start, stop);

    // ---------------- Kernel launch ----------------
    int threads = 256;
    int blocks  = (N + threads - 1) / threads;
    hipEventRecord(start);
    saxpy<<<blocks, threads>>>(N, 3.0f, d_x, d_y);
    hipEventRecord(stop);  hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_ms, start, stop);

    // ---------------- Device ➜ Host copy ----------------
    hipEventRecord(start);
    hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop);  hipEventSynchronize(stop);
    hipEventElapsedTime(&d2h_ms, start, stop);


    printf("N=%d  H2D=%.3f ms  kernel=%.3f ms  D2H=%.3f ms\n",
           N, h2d_ms, kernel_ms, d2h_ms);

    // Quick correctness check
    float max_err = 0.0f;
    for (int i = 0; i < N; ++i) max_err = fmax(max_err, fabs(h_y[i] - 5.0f));
    printf("Max error: %f\n", max_err);

    hipFree(d_x); hipFree(d_y); free(h_x); free(h_y);
    return 0;
}

