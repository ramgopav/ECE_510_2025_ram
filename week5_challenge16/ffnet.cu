#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#define CHECK(x) do{auto e=(x); if(e!=HIPBLAS_STATUS_SUCCESS){printf("cuBLAS %d\n",e); return 1;}}while(0)
__global__ void relu(float* x,int n){int i=blockIdx.x*blockDim.x+threadIdx.x; if(i<n&&x[i]<0) x[i]=0;}
int main(){
    const int NI=4,NH=5,NO=1; float hx[NI],W1[NH*NI],b1[NH],W2[NO*NH],b2[NO];
    FILE*fp=fopen("weights.bin","rb");
    fread(hx,4,NI,fp); fread(W1,4,NH*NI,fp); fread(b1,4,NH,fp); fread(W2,4,NO*NH,fp); fread(b2,4,NO,fp); fclose(fp);
    float *dx,*dW1,*db1,*dW2,*db2; hipMalloc(&dx,16); hipMalloc(&dW1,4*NH*NI); hipMalloc(&db1,4*NH);
    hipMalloc(&dW2,4*NO*NH); hipMalloc(&db2,4*NO);
    hipMemcpy(dx,hx,16,hipMemcpyHostToDevice);
    hipMemcpy(dW1,W1,4*NH*NI,hipMemcpyHostToDevice);
    hipMemcpy(db1,b1,4*NH,hipMemcpyHostToDevice);
    hipMemcpy(dW2,W2,4*NO*NH,hipMemcpyHostToDevice);
    hipMemcpy(db2,b2,4*NO,hipMemcpyHostToDevice);
    hipblasHandle_t h; CHECK(hipblasCreate(&h));
    const float a=1.f,b=1.f;
    for(int i=0;i<5;++i) CHECK(hipblasSgemv(h,HIPBLAS_OP_T,NI,NH,&a,dW1,NI,dx,1,&b,db1,1));
    hipMemcpy(db1,b1,4*NH,hipMemcpyHostToDevice);
    hipMemcpy(db2,b2,4*NO,hipMemcpyHostToDevice);
    hipEvent_t t0,t1; hipEventCreate(&t0); hipEventCreate(&t1); hipEventRecord(t0);
    CHECK(hipblasSgemv(h,HIPBLAS_OP_T,NI,NH,&a,dW1,NI,dx,1,&b,db1,1));
    relu<<<1,32>>>(db1,NH);
    CHECK(hipblasSgemv(h,HIPBLAS_OP_T,NH,NO,&a,dW2,NH,db1,1,&b,db2,1));
    hipEventRecord(t1); hipEventSynchronize(t1); float ms; hipEventElapsedTime(&ms,t0,t1);
    float out; hipMemcpy(&out,db2,4,hipMemcpyDeviceToHost);
    printf("CUDA out %.6f | %.3f µs\n",out,ms*1000);
}
